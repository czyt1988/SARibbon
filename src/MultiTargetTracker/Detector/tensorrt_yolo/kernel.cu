
#include <hip/hip_runtime.h>

#include <stdint.h>
#include <stdio.h>
#include <string.h>

inline __device__ float sigmoidGPU(const float& x) { return 1.0f / (1.0f + __expf(-x)); }

__global__ void gpuYoloLayerV3(const float* input, float* output, const uint32_t grid_h_,
								const uint32_t grid_w_, const uint32_t numOutputClasses,
                               const uint32_t numBBoxes)
{
    uint32_t x_id = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y_id = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t z_id = blockIdx.z * blockDim.z + threadIdx.z;

    if ((x_id >= grid_w_) || (y_id >= grid_h_) || (z_id >= numBBoxes))
    {
        return;
    }

    const int numGridCells = grid_h_ * grid_w_;
    const int bbindex = y_id * grid_w_ + x_id;

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 0)]
        = sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 0)]);

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 1)]
        = sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 1)]);

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)]
        = __expf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)]);

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)]
        = __expf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)]);

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)]
        = sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)]);

    for (uint32_t i = 0; i < numOutputClasses; ++i)
    {
        output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))]
            = sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))]);
    }
}

hipError_t cudaYoloLayerV3(const void* input, void* output, const uint32_t& batchSize, 
							const uint32_t& n_grid_h_,const uint32_t& n_grid_w_,
                            const uint32_t& numOutputClasses, const uint32_t& numBBoxes,
                            uint64_t outputSize, hipStream_t stream)
{
    dim3 threads_per_block(16, 16, 4);
    dim3 number_of_blocks((n_grid_w_ / threads_per_block.x) + 1,
                          (n_grid_h_ / threads_per_block.y) + 1,
                          (numBBoxes / threads_per_block.z) + 1);
    for (int batch = 0; batch < batchSize; ++batch)
    {
        gpuYoloLayerV3<<<number_of_blocks, threads_per_block, 0, stream>>>(
            reinterpret_cast<const float*>(input) + (batch * outputSize),
            reinterpret_cast<float*>(output) + (batch * outputSize), n_grid_h_, n_grid_w_, numOutputClasses,
            numBBoxes);
    }
    return hipGetLastError();
}
