#include "hip/hip_runtime.h"
#include <cmath>
#include <stdio.h>
#include <cassert>
#include <iostream>
#include "mish.h"

namespace nvinfer1
{
    MishPlugin::MishPlugin()
    {
    }

    MishPlugin::~MishPlugin()
    {
    }

    // create the plugin at runtime from a byte stream
    MishPlugin::MishPlugin(const void* data, size_t length)
    {
        assert(length == sizeof(input_size_));
        input_size_ = *reinterpret_cast<const int*>(data);
    }

    void MishPlugin::serialize(void* buffer) const
    {
        *reinterpret_cast<int*>(buffer) = input_size_;
    }

    size_t MishPlugin::getSerializationSize() const
    {  
        return sizeof(input_size_);
    }

    int MishPlugin::initialize()
    { 
        return 0;
    }

    Dims MishPlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims)
    {
        assert(nbInputDims == 1);
        assert(index == 0);
        input_size_ = inputs[0].d[0] * inputs[0].d[1] * inputs[0].d[2];
        // Output dimensions
        return Dims3(inputs[0].d[0], inputs[0].d[1], inputs[0].d[2]);
    }

    // Set plugin namespace
    void MishPlugin::setPluginNamespace(const char* pluginNamespace)
    {
        mPluginNamespace = pluginNamespace;
    }

    const char* MishPlugin::getPluginNamespace() const
    {
        return mPluginNamespace;
    }

    // Return the DataType of the plugin output at the requested index
    DataType MishPlugin::getOutputDataType(int index, const nvinfer1::DataType* inputTypes, int nbInputs) const
    {
        return DataType::kFLOAT;
    }

    // Return true if output tensor is broadcast across a batch.
    bool MishPlugin::isOutputBroadcastAcrossBatch(int outputIndex, const bool* inputIsBroadcasted, int nbInputs) const
    {
        return false;
    }

    // Return true if plugin can use input that is broadcast across batch without replication.
    bool MishPlugin::canBroadcastInputAcrossBatch(int inputIndex) const
    {
        return false;
    }

    void MishPlugin::configurePlugin(const PluginTensorDesc* in, int nbInput, const PluginTensorDesc* out, int nbOutput)
    {
    }

    // Attach the plugin object to an execution context and grant the plugin the access to some context resource.
    void MishPlugin::attachToContext(cudnnContext* cudnnContext, cublasContext* cublasContext, IGpuAllocator* gpuAllocator)
    {
    }

    // Detach the plugin object from its execution context.
    void MishPlugin::detachFromContext() {}

    const char* MishPlugin::getPluginType() const
    {
        return "Mish_TRT";
    }

    const char* MishPlugin::getPluginVersion() const
    {
        return "1";
    }

    void MishPlugin::destroy()
    {
        delete this;
    }

    // Clone the plugin
    IPluginV2IOExt* MishPlugin::clone() const
    {
        MishPlugin *p = new MishPlugin();
        p->input_size_ = input_size_;
        p->setPluginNamespace(mPluginNamespace);
        return p;
    }

    __device__ float tanh_activate_kernel(float x){return (2/(1 + expf(-2*x)) - 1);}

    __device__ float softplus_kernel(float x, float threshold = 20) {
        if (x > threshold) return x;                // too large
        else if (x < -threshold) return expf(x);    // too small
        return logf(expf(x) + 1);
    }

    __global__ void mish_kernel(const float *input, float *output, int num_elem) {

        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= num_elem) return;

        //float t = exp(input[idx]);
        //if (input[idx] > 20.0) {
        //    t *= t;
        //    output[idx] = (t - 1.0) / (t + 1.0);
        //} else {
        //    float tt = t * t;
        //    output[idx] = (tt + 2.0 * t) / (tt + 2.0 * t + 2.0);
        //}
        //output[idx] *= input[idx];
        output[idx] = input[idx] * tanh_activate_kernel(softplus_kernel(input[idx]));
    }

    void MishPlugin::forwardGpu(const float *const * inputs, float* output, hipStream_t stream, int batchSize) {
        int block_size = thread_count_;
        int grid_size = (input_size_ * batchSize + block_size - 1) / block_size;
        mish_kernel<<<grid_size, block_size>>>(inputs[0], output, input_size_ * batchSize);
    }

    int MishPlugin::enqueue(int batchSize, const void*const * inputs, void** outputs, void* workspace, hipStream_t stream)
    {
        //assert(batchSize == 1);
        //GPU
        //CUDA_CHECK(hipStreamSynchronize(stream));
        forwardGpu((const float *const *)inputs, (float*)outputs[0], stream, batchSize);
        return 0;
    }

    PluginFieldCollection MishPluginCreator::mFC{};
    std::vector<PluginField> MishPluginCreator::mPluginAttributes;

    MishPluginCreator::MishPluginCreator()
    {
        mPluginAttributes.clear();

        mFC.nbFields = mPluginAttributes.size();
        mFC.fields = mPluginAttributes.data();
    }

    const char* MishPluginCreator::getPluginName() const
    {
            return "Mish_TRT";
    }

    const char* MishPluginCreator::getPluginVersion() const
    {
            return "1";
    }

    const PluginFieldCollection* MishPluginCreator::getFieldNames()
    {
            return &mFC;
    }

    IPluginV2IOExt* MishPluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc)
    {
        MishPlugin* obj = new MishPlugin();
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

    IPluginV2IOExt* MishPluginCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength)
    {
        // This object will be deleted when the network is destroyed, which will
        // call MishPlugin::destroy()
        MishPlugin* obj = new MishPlugin(serialData, serialLength);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

}

